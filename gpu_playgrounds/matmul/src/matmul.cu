#include <stdio.h>
#include <iostream>
#include <random>
#include <assert.h>
#include <iomanip>
#include <chrono>
#include <hip/hip_runtime.h>

#define DEBUG 0

#define M 2000
#define N 1000
#define K 1500

#define TILE_WIDTH 16

std::random_device rd;
std::mt19937 gen(rd());
std::uniform_real_distribution<float> dis(0.0, 100.0);

struct Matrix {
    std::string name;
    float* m;
    int numBytes;
    int numRows;
    int numCols;

    Matrix(std::string n, int r, int c, bool populate) : name(std::move(n)), numRows(r), numCols(c) {
        int numElements = numRows * numCols;
        m = (float*) malloc(numElements * sizeof(float));
        if (populate) {
            for (int i = 0; i < numElements; ++i) {
                m[i] = dis(gen);
            }
        }
        numBytes = numElements * sizeof(float);
    }

    __host__ void print() {
        std::cout << "Matrix " << name << ":" << std::endl;
        for (int i = 0; i < numRows; ++i) {
            std::cout << "\t";
            for (int j = 0; j < numCols; ++j) {
                std::cout << m[i * numCols + j] << " ";
            }
            std::cout << std::endl;
        }
        std::cout << std::endl;
    }
};

__host__ bool compare(const Matrix& C, const Matrix& CGPU) {
    assert(C.numRows == CGPU.numRows && C.numCols == CGPU.numCols);
    for (int r = 0; r < C.numRows; ++r) {
        for (int c = 0; c < C.numCols; ++c) {
            int idx = r * C.numCols + c;
            if (std::abs(C.m[idx] - CGPU.m[idx]) > 5) {
                std::cout << std::setprecision(10) 
                    << "Failure on r = " << r
                    << ", c = " << c
                    << ": " << C.m[idx] << " != " << CGPU.m[idx] << std::endl;
                return false;
            }
        }
    }
    return true;
}

__host__ void hostMatMul(const Matrix& A, const Matrix& B, const Matrix& C) {
    auto start = std::chrono::high_resolution_clock::now();
    for (int r = 0; r < M; ++r) {
        for (int c = 0; c < K; ++c) {
            float sum = 0.0;
            for (int i = 0; i < N; ++i) {
                sum += A.m[r * N + i] * B.m[c + i * K];
            }
            C.m[r * K + c] = sum;
        }
    }
    auto stop = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(stop - start);
    std::cout << "Time for CPU matmul execution: " << duration.count() << " milliseconds." << std::endl;
}

__global__ void kernelMatMul(int m, int n, int k, float* A, float* B, float* C) {
    __shared__ float ds_A[TILE_WIDTH][TILE_WIDTH];
    __shared__ float ds_B[TILE_WIDTH][TILE_WIDTH];

    int bx = blockIdx.x; int by = blockIdx.y;
    int tx = threadIdx.x; int ty = threadIdx.y;

    int row = by * blockDim.y + ty;
    int col = bx * blockDim.x + tx;
    float CVal = 0;

    for (int tile = 0; tile < (n - 1) / TILE_WIDTH + 1; ++tile) {
        // Load into tile A
        if (row < m && tile * TILE_WIDTH + tx < n) {
            ds_A[ty][tx] = A[row * n + (tile * TILE_WIDTH + tx)];
        } else {
            ds_A[ty][tx] = 0;
        }

        // Load into tile B
        if (tile * TILE_WIDTH + ty < n && col < k) {
            ds_B[ty][tx] = B[(ty + tile * TILE_WIDTH) * k + col];
        } else {
            ds_B[ty][tx] = 0;
        }

        __syncthreads();

        for (int i = 0; i < TILE_WIDTH; ++i) {
            CVal += ds_A[ty][i] * ds_B[i][tx];
        }
        __syncthreads();
    }

    if (row < m && col < k) {
        C[row * k + col] = CVal;
    }
}

__host__ void gpuMatMul(Matrix& A, Matrix& B, Matrix& C) {
    hipEvent_t start, stop;
    hipEventCreate(&start); 
    hipEventCreate(&stop);

    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, A.numBytes);
    hipMalloc(&d_B, B.numBytes);
    hipMalloc(&d_C, C.numBytes);
    hipMemcpy(d_A, A.m, A.numBytes, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B.m, B.numBytes, hipMemcpyHostToDevice);

    // Execute kernel
    dim3 dimGrid((K - 1)/TILE_WIDTH + 1, (M - 1)/TILE_WIDTH + 1, 1);
    dim3 dimBlock(TILE_WIDTH, TILE_WIDTH, 1);

    hipEventRecord(start);
    kernelMatMul<<<dimGrid, dimBlock>>>(M, N, K, d_A, d_B, d_C);
    hipEventRecord(stop);

    hipEventSynchronize(stop);

    float ms = 0;
    hipEventElapsedTime(&ms, start, stop);
    std::cout << "Time for GPU matmul execution: " << ms << " milliseconds." << std::endl;
    hipEventDestroy(start);
    hipEventDestroy(stop);

    // Copy results back to host
    hipMemcpy(C.m, d_C, C.numBytes, hipMemcpyDeviceToHost);
}

int main() {
    // Generate random A and B matrices
    Matrix A("A", M, N, true);
    Matrix B("B", N, K, true);
    Matrix CCPU("C CPU", M, K, false);
    Matrix CGPU("C GPU", M, K, false);

    if constexpr(DEBUG) {
        A.print();
        B.print();
    }

    hostMatMul(A, B, CCPU);
    if constexpr(DEBUG) {
        CCPU.print();
    }

    gpuMatMul(A, B, CGPU);
    if constexpr(DEBUG) {
        CGPU.print();
    }

    // Check that the implementation is correct
    if (compare(CCPU, CGPU) == true) {
        std::cout << "Success!" << std::endl;
    }
}
