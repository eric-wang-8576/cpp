
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include <random>
#include <assert.h>
#include <iomanip>
#include <chrono>

#define DEBUG 0

#define M 2000
#define N 1800
#define K 1900

#define TILE_WIDTH 16

std::random_device rd;
std::mt19937 gen(rd());
std::uniform_real_distribution<float> dis(0.0, 100.0);

struct Matrix {
    std::string name;
    float* m;
    int numBytes;
    int numRows;
    int numCols;

    Matrix(std::string n, int r, int c, bool populate) : name(std::move(n)), numRows(r), numCols(c) {
        int numElements = numRows * numCols;
        m = (float*) malloc(numElements * sizeof(float));
        if (populate) {
            for (int i = 0; i < numElements; ++i) {
                m[i] = dis(gen);
            }
        }
        numBytes = numElements * sizeof(float);
    }

    __host__ void print() {
        std::cout << "Matrix " << name << ":" << std::endl;
        for (int i = 0; i < numRows; ++i) {
            std::cout << "\t";
            for (int j = 0; j < numCols; ++j) {
                std::cout << m[i * numCols + j] << " ";
            }
            std::cout << std::endl;
        }
        std::cout << std::endl;
    }
};

__host__ void hostMatMul(const Matrix& A, const Matrix& B, const Matrix& C) {
    for (int r = 0; r < M; ++r) {
        for (int c = 0; c < K; ++c) {
            float sum = 0.0;
            for (int i = 0; i < N; ++i) {
                sum += A.m[r * N + i] * B.m[c + i * K];
            }
            C.m[r * K + c] = sum;
        }
    }
}

__host__ bool compare(const Matrix& C, const Matrix& CGPU) {
    assert(C.numRows == CGPU.numRows && C.numCols == CGPU.numCols);
    for (int r = 0; r < C.numRows; ++r) {
        for (int c = 0; c < C.numCols; ++c) {
            int idx = r * C.numCols + c;
            if (std::abs(C.m[idx] - CGPU.m[idx]) > 5) {
                std::cout << std::setprecision(10) 
                    << "Failure on r = " << r
                    << ", c = " << c
                    << ": " << C.m[idx] << " != " << CGPU.m[idx] << std::endl;
                return false;
            }
        }
    }
    return true;
}

__global__ void kernelMatMul(int m, int n, int k, float* A, float* B, float* C) {
    int r = blockIdx.x * blockDim.x + threadIdx.x;
    int c = blockIdx.y * blockDim.y + threadIdx.y;

    if (r < m && c < k) {
        float sum = 0.0;
        for (int i = 0; i < N; ++i) {
            sum += A[r * n + i] * B[c + i * k];
        }
        C[r * k + c] = sum;
    }
}

int main() {
    // Generate random A and B matrices
    Matrix A("A", M, N, true);
    Matrix B("B", N, K, true);
    Matrix C("C Correct", M, K, false);
    Matrix CGPU("C Generated", M, K, false);

    if constexpr(DEBUG) {
        A.print();
        B.print();
    }

    // Generate the correct matrix C
    auto start = std::chrono::steady_clock::now();

    hostMatMul(A, B, C);

    auto end = std::chrono::steady_clock::now();
    auto elapsed = std::chrono::duration_cast<std::chrono::milliseconds>(end - start);
    std::cout << "Time for CPU execution: " << elapsed.count() << " milliseconds." << std::endl;
    if constexpr(DEBUG) {
        C.print();
    }

    // Allocate row-major order matrices on the device
    start = std::chrono::steady_clock::now();

    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, A.numBytes);
    hipMalloc(&d_B, B.numBytes);
    hipMalloc(&d_C, C.numBytes);
    hipMemcpy(d_A, A.m, A.numBytes, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B.m, B.numBytes, hipMemcpyHostToDevice);

    // Execute kernel
    dim3 dimGrid((M - 1)/TILE_WIDTH + 1, (K - 1)/TILE_WIDTH + 1, 1);
    dim3 dimBlock(TILE_WIDTH, TILE_WIDTH, 1);

    kernelMatMul<<<dimGrid, dimBlock>>>(M, N, K, d_A, d_B, d_C);
    
    // Copy results back to host
    hipMemcpy(CGPU.m, d_C, C.numBytes, hipMemcpyDeviceToHost);
    if constexpr(DEBUG) {
        CGPU.print();
    }

    end = std::chrono::steady_clock::now();
    elapsed = std::chrono::duration_cast<std::chrono::milliseconds>(end - start);
    std::cout << "Time for GPU execution: " << elapsed.count() << " milliseconds." << std::endl;



    // Check that the implementation is correct
    if (compare(C, CGPU) == true) {
        std::cout << "Success!" << std::endl;
    }
}
