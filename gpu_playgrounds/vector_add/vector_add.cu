
#include <hip/hip_runtime.h>
#include <stdio.h>

// CUDA Kernel for Vector Addition
__global__ void vecAdd(float *A, float *B, float *C, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
        C[i] = A[i] + B[i];
    }
}

int main() {
    int N = 1024; // Size of vectors
    float *A, *B, *C; // Host vectors
    float *d_A, *d_B, *d_C; // Device (GPU) vectors

    // Allocate memory on the host
    A = (float *)malloc(N * sizeof(float));
    B = (float *)malloc(N * sizeof(float));
    C = (float *)malloc(N * sizeof(float));

    // Initialize vectors on the host
    for(int i = 0; i < N; i++) {
        A[i] = i * 5;
        B[i] = i * 2;
    }

    // Allocate memory on the device
    hipMalloc(&d_A, N * sizeof(float));
    hipMalloc(&d_B, N * sizeof(float));
    hipMalloc(&d_C, N * sizeof(float));

    // Copy host vectors to device
    hipMemcpy(d_A, A, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, N * sizeof(float), hipMemcpyHostToDevice);

    // Execute the vector addition kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    vecAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);

    // Copy result back to host
    hipMemcpy(C, d_C, N * sizeof(float), hipMemcpyDeviceToHost);

    // Display the result
    for(int i = 0; i < 10; i++) {
        printf("C[%d] = %f\n", i, C[i]);
    }

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // Free host memory
    free(A);
    free(B);
    free(C);

    return 0;
}
